#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "boost/format.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

using boost::format;

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithPerClassLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  if (normalize_) {
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);
    loss = count==0? 0 : loss / count;
    //loss /= count;
  } else {
    loss = outer_num_==0? 0 : loss / outer_num_;
    //loss /= outer_num_;
  }
  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts,
          const Dtype* lr_mult) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    // we assume spatial_dim to be 1 most of the time (W*H)
    // so we shall assume n indexes sample, s is always 0
    // HYQ we add LR scaling directly in here.
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      //if (dim==29) // print only for category, alright
      //  printf("n=%i, dim=%i, spatial_dim=%i, label_value=%i, lr_mult=%f\n", n, dim, spatial_dim, label_value, lr_mult[n]);
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      // HYQ
      bottom_diff[n * dim + label_value * spatial_dim + s] *= lr_mult[n];
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithPerClassLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const Dtype* label_cpu = bottom[1]->cpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    //LOG(INFO) << "nthreads=" << nthreads;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    //HYQ
    // before we call softmax gpu backprop, we need to load each sample's label
    // and lr_mult into lr_mult_
    Dtype* lr_mult_cpu = lr_mult_.mutable_cpu_data();

    
    for(int i = 0; i<outer_num_; ++i){
      CHECK_GE(label_cpu[i],0);
      CHECK_LT(label_cpu[i],num_classes_);
      lr_mult_cpu[i] = class_lrmults_[label_cpu[i]];
    }
    lr_mult_.gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts,
        lr_mult_.mutable_gpu_data());
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(nthreads, counts, &count);
      caffe_gpu_scal(prob_.count(), 
                     count==0? 0 : loss_weight / count, //HYQ
                     bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), 
                     loss_weight / outer_num_, 
                     bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithPerClassLossLayer);

}  // namespace caffe
